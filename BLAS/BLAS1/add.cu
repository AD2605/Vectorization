#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>

// CUDA kernel. Each thread takes care of one element of c

__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    uint id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

__global__ void matrixAdd(double * a, double * b, double * c,int width, int height ){
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i< width and j < height){
        *(c + i*width + j) = *(a + i*width + j) + *(b + i*width + j);
    }

}

int main( int argc, char* argv[] )
{
    int width = 4096;
    int height = 8192;
    double count = 1.0;

    dim3 threads(512, 512);
    dim3 groups(int(ceil(width*height*sizeof(double )/threads.x)), int(ceil(width*height* sizeof(double )/threads.y)));

    dim3 threads_1D(1024);
    dim3 groups_1D(int(ceil(width*height*sizeof(double )/threads.x)));

    double * host_A, *host_B,*host_C, *host_C1D;
    double* device_A, *device_B, *device_C, *device_C1D;

    host_A = (double *)malloc(width * height * sizeof(double ));
    host_B = (double *)malloc(width * height * sizeof(double ));
    host_C = (double *)malloc(width * height * sizeof(double ));
    host_C1D = (double *)malloc(width * height * sizeof(double ));

    for(int i = 0; i<width; i++){
        for (int j = 0;j<height;j++){
            *(host_A + i*width + j) = *(host_B + i*width + j) = ++count;
        }
    }

    hipMalloc((&device_A), width*height* sizeof(double ));
    hipMalloc((&device_B), width*height* sizeof(double ));
    hipMalloc((&device_C), width*height* sizeof(double ));
    hipMalloc((&device_C1D), width*height* sizeof(double ));

    hipMemcpy(device_A, host_A, width*height* sizeof(double ), hipMemcpyHostToDevice);
    hipMemcpy(device_B, host_B, width*height* sizeof(double ), hipMemcpyHostToDevice);

    matrixAdd<<<groups, threads>>>(device_A, device_B, device_C, width);
    vecAdd<<<groups_1D, threads_1D>>>(device_A, device_B, device_C1D);

    hipMemcpy(host_C, device_C, width*height* sizeof(double ), hipMemcpyDeviceToHost);

    for(int i = 0; i<width; i++){
        for (int j = 0;j<height;j++){
            printf("%f ", *(host_A + i*width + j));
            printf("%s", "  ");
        }
    }

}