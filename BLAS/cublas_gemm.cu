
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <ctime>
#include <iostream>

void gpu_cublas(const float* A, const float* B, float* C, int m, int n, int k)
{
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m,n,k, alpha, A, lda, B, ldb, beta, C, ldc);
    hipblasDestroy(handle);
}

int main()
{
    int nr_rows_A =3,
        nr_cols_A=3,
        nr_rows_B=3,
        nr_cols_B=3,
        nr_rows_C=3,
        nr_cols_C=3;

    float *d_A, *d_B, *d_C;
    auto *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
    auto *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    auto *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

    hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
    hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
    hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, (unsigned long long ) clock());
    hiprandGenerateUniform(generator, h_A, nr_rows_A * nr_cols_A);
    hiprandGenerateUniform(generator, h_B, nr_rows_B*nr_cols_B);
    std::cout<<"GENERATED MATRICES";
    //cudaMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),cudaMemcpyHostToDevice);
    //cudaMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),cudaMemcpyHostToDevice);
    gpu_cublas(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_rows_B);
    hipMemcpy(h_C,d_C, nr_rows_C*nr_cols_C*sizeof(float), hipMemcpyDeviceToHost);
    std::cout<<"C = "<<std::endl;
    for (int i = 0; i <nr_rows_A ; ++i) {
        for (int j = 0;j<nr_rows_A; ++j){
            std::cout << h_C[j * nr_rows_A + i] << " ";
        }
        std::cout<<std::endl;
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

}